#include "utils.h"

void checkCudaError() {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);
        fprintf(stderr, "code: %d, reason: %s\n", error, hipGetErrorString(error));
    }
}